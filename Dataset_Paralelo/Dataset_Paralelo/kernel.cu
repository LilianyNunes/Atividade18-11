﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cstring>

// Função para ler uma imagem BMP (500x500)
void readImageBMP(const std::string& filename, std::vector<unsigned char>& data, int& width, int& height) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Erro ao abrir o arquivo BMP." << std::endl;
        return;
    }

    // Cabeçalho de 54 bytes (14 + 40)
    unsigned char header[54];
    file.read(reinterpret_cast<char*>(header), 54);

    // Largura e altura
    width = *reinterpret_cast<int*>(&header[18]);
    height = *reinterpret_cast<int*>(&header[22]);

    // Verificando se a imagem é 24 bits por pixel
    int bitsPerPixel = *reinterpret_cast<short*>(&header[28]);
    if (bitsPerPixel != 24) {
        std::cerr << "A imagem deve ser 24 bits por pixel." << std::endl;
        return;
    }

    // Tamanho da linha de dados, considerando o padding
    int padding = (4 - (width * 3) % 4) % 4;  // 3 porque cada pixel tem 3 bytes (RGB)
    int rowSize = width * 3 + padding;

    // Lendo os dados de pixel
    data.resize(rowSize * height);
    for (int y = height - 1; y >= 0; --y) {
        file.read(reinterpret_cast<char*>(&data[y * rowSize]), rowSize);
    }

    file.close();
}

// Função para salvar a imagem em formato BMP
void saveImageBMP(const std::string& filename, const std::vector<unsigned char>& data, int width, int height) {
    int padding = (4 - (width * 3) % 4) % 4; // Preenchimento para alinhar as linhas a múltiplos de 4 bytes
    int rowSize = width * 3 + padding;
    int dataSize = rowSize * height;

    // Cabeçalho do arquivo BMP (14 bytes)
    unsigned char fileHeader[14] = {
        'B', 'M',                           // Identificação do arquivo BMP
        0, 0, 0, 0,                         // Tamanho total do arquivo (atualizado depois)
        0, 0,                               // Reservado
        0, 0,                               // Reservado
        54, 0, 0, 0                         // Offset para os dados de pixel
    };

    // Cabeçalho da informação BMP (40 bytes)
    unsigned char infoHeader[40] = {
        40, 0, 0, 0,                        // Tamanho deste cabeçalho
        0, 0, 0, 0,                         // Largura da imagem (atualizado depois)
        0, 0, 0, 0,                         // Altura da imagem (atualizado depois)
        1, 0,                               // Planos (sempre 1)
        24, 0,                              // Bits por pixel (24 para RGB)
        0, 0, 0, 0,                         // Nenhuma compressão
        0, 0, 0, 0,                         // Tamanho dos dados de imagem (atualizado depois)
        0, 0, 0, 0,                         // Resolução horizontal (não usada)
        0, 0, 0, 0,                         // Resolução vertical (não usada)
        0, 0, 0, 0,                         // Número de cores na paleta
        0, 0, 0, 0                          // Todas as cores são importantes
    };

    // Atualizando os cabeçalhos com os tamanhos corretos
    int fileSize = 54 + dataSize;
    std::memcpy(&fileHeader[2], &fileSize, 4);
    std::memcpy(&infoHeader[4], &width, 4);
    std::memcpy(&infoHeader[8], &height, 4);
    std::memcpy(&infoHeader[20], &dataSize, 4);

    // Abrindo o arquivo para escrita
    std::ofstream file(filename, std::ios::out | std::ios::binary);
    if (!file) {
        std::cerr << "Erro ao abrir arquivo para escrita: " << filename << std::endl;
        return;
    }

    // Gravando cabeçalhos
    file.write(reinterpret_cast<const char*>(fileHeader), sizeof(fileHeader));
    file.write(reinterpret_cast<const char*>(infoHeader), sizeof(infoHeader));

    // Gravando os dados dos pixels
    for (int y = height - 1; y >= 0; --y) { // BMP começa do canto inferior
        file.write(reinterpret_cast<const char*>(&data[y * rowSize]), width * 3);
        file.write("\0\0\0", padding); // Adicionando o padding
    }

    file.close();
    std::cout << "Imagem salva como BMP: " << filename << std::endl;
}

// Função para converter uma imagem para tons de cinza
void convertToGray(std::vector<unsigned char>& data, int width, int height) {
    int padding = (4 - (width * 3) % 4) % 4; // Preenchimento para alinhar as linhas a múltiplos de 4 bytes
    int rowSize = width * 3 + padding;

    for (int y = height - 1; y >= 0; --y) {
        for (int x = 0; x < width; ++x) {
            // Índices para os canais RGB
            int pixelIdx = y * rowSize + x * 3;

            unsigned char r = data[pixelIdx + 2]; // Vermelho
            unsigned char g = data[pixelIdx + 1]; // Verde
            unsigned char b = data[pixelIdx + 0]; // Azul

            // Calculando a intensidade de cinza usando a fórmula de luminosidade
            unsigned char gray = static_cast<unsigned char>(r * 0.298 + g * 0.587 + b * 0.114);

            // Substituindo os valores RGB pelo valor de cinza
            data[pixelIdx + 2] = gray; // Vermelho
            data[pixelIdx + 1] = gray; // Verde
            data[pixelIdx + 0] = gray; // Azul
        }
    }
}

int main() {
    std::string inputFilename = "teste.bmp";
    std::string outputFilename = "output.bmp";

    // Dados da imagem
    int width, height;
    std::vector<unsigned char> imageData;

    // Lendo a imagem
    readImageBMP(inputFilename, imageData, width, height);

    // Convertendo a imagem para tons de cinza
    convertToGray(imageData, width, height);

    // Salvando a imagem convertida
    saveImageBMP(outputFilename, imageData, width, height);

    return 0;
}
