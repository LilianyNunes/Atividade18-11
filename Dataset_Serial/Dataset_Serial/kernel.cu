﻿#include <iostream>

#include <fstream>

#include <vector>

#include <cmath>

#include <string>

#include <chrono> // Para medir o tempo de execução

#include <hip/hip_runtime.h>



#pragma pack(push, 1)

struct BMPHeader {

    char signature[2];

    uint32_t fileSize;

    uint32_t reserved;

    uint32_t dataOffset;

    uint32_t headerSize;

    int32_t width;

    int32_t height;

    uint16_t colorPlanes;

    uint16_t bitsPerPixel;

    uint32_t compression;

    uint32_t dataSize;

    int32_t hResolution;

    int32_t vResolution;

    uint32_t colors;

    uint32_t importantColors;

};

#pragma pack(pop)



bool saveBMP(const std::string& filename, const std::vector<uint8_t>& imageData, int width, int height) {

    std::ofstream file(filename, std::ios::binary);

    if (!file) {

        std::cerr << "Erro ao salvar o arquivo." << std::endl;

        return false;

    }



    int rowSize = ((width * 3 + 3) & ~3);

    int paddedDataSize = rowSize * height;



    BMPHeader header;

    header.signature[0] = 'B';

    header.signature[1] = 'M';

    header.fileSize = sizeof(BMPHeader) + paddedDataSize;

    header.reserved = 0;

    header.dataOffset = sizeof(BMPHeader);

    header.headerSize = 40;

    header.width = width;

    header.height = height;

    header.colorPlanes = 1;

    header.bitsPerPixel = 24;

    header.compression = 0;

    header.dataSize = paddedDataSize;

    header.hResolution = 2835;

    header.vResolution = 2835;

    header.colors = 0;

    header.importantColors = 0;



    file.write(reinterpret_cast<char*>(&header), sizeof(header));

    std::vector<uint8_t> padding(rowSize - width * 3, 0);

    for (int y = 0; y < height; ++y) {

        file.write(reinterpret_cast<const char*>(&imageData[y * width * 3]), width * 3);

        file.write(reinterpret_cast<const char*>(padding.data()), padding.size());

    }

    file.close();

    return true;

}



bool loadBMP(const std::string& filename, std::vector<uint8_t>& imageData, int& width, int& height) {

    std::ifstream file(filename, std::ios::binary);

    if (!file) {

        std::cerr << "Erro ao abrir o arquivo." << std::endl;

        return false;

    }



    BMPHeader header;

    file.read(reinterpret_cast<char*>(&header), sizeof(header));

    if (header.signature[0] != 'B' || header.signature[1] != 'M') {

        std::cerr << "Não é um arquivo BMP válido." << std::endl;

        return false;

    }



    width = header.width;

    height = header.height;

    int rowSize = ((width * 3 + 3) & ~3);



    imageData.resize(width * height * 3);

    std::vector<uint8_t> padding(rowSize - width * 3);



    for (int y = 0; y < height; ++y) {

        file.read(reinterpret_cast<char*>(&imageData[y * width * 3]), width * 3);

        file.read(reinterpret_cast<char*>(padding.data()), padding.size());

    }

    file.close();

    return true;

}



void convertToGrayscale(std::vector<uint8_t>& imageData, int width, int height) {

    for (int i = 0; i < width * height; ++i) {

        int idx = i * 3;

        uint8_t r = imageData[idx];

        uint8_t g = imageData[idx + 1];

        uint8_t b = imageData[idx + 2];

        uint8_t gray = static_cast<uint8_t>(r * 0.298 + g * 0.587 + b * 0.114);

        imageData[idx] = gray;

        imageData[idx + 1] = gray;

        imageData[idx + 2] = gray;

    }

}



void measurePerformance(int numImages, const std::string& inputFilename, const std::string& outputFilename) {

    int width, height;

    std::vector<uint8_t> imageData;



    if (!loadBMP(inputFilename, imageData, width, height)) {

        std::cerr << "Erro ao carregar a imagem de entrada." << std::endl;

        return;

    }



    auto start = std::chrono::high_resolution_clock::now();



    for (int i = 0; i < numImages; ++i) {

        std::vector<uint8_t> imageCopy = imageData;

        convertToGrayscale(imageCopy, width, height);

        saveBMP(outputFilename + std::to_string(i) + ".bmp", imageCopy, width, height);

    }



    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;



    std::cout << "Tempo para processar " << numImages << " imagens: " << elapsed.count() << " segundos." << std::endl;

}



int main() {

    std::string inputFilename = "teste.bmp";

    std::string outputFilename = "imagem cinza_";



    std::cout << "Metricas de tempo de execucao:" << std::endl;



    measurePerformance(1, inputFilename, outputFilename);

    measurePerformance(10, inputFilename, outputFilename);

    measurePerformance(100, inputFilename, outputFilename);

    measurePerformance(1000, inputFilename, outputFilename);



    return 0;

}